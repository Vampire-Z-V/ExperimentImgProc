#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


__global__ void collection(
	int srcWidth, int srcHeight,
	bool isRGB,
	int * srcR, int * srcG, int * srcB,
	double * totalR, double * totalG, double * totalB
)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < srcWidth && y < srcHeight)
	{
		int index = x + y * srcWidth;
		if (isRGB)
		{
			*totalR += srcR[index];
			*totalG += srcG[index];
		}
		*totalB += srcB[index];
	}
}

extern "C"
void collectRGB(
	int srcWidth, int srcHeight,
	bool isRGB,
	int * srcR, int * srcG, int * srcB,
	double * totalR, double * totalG, double * totalB
)
{
	int * srcR_in;
	int * srcG_in;
	int * srcB_in;
	double *totalR_out;
	double *totalG_out;
	double *totalB_out;

	int srcSize = srcWidth * srcHeight;

	dim3 dimBlock(32, 32);
	dim3 dimGrid(
		(srcWidth + dimBlock.x - 1) / dimBlock.x,
		(srcHeight + dimBlock.y - 1) / dimBlock.y
	);

	if (isRGB)
	{
		checkCudaErrors(hipMalloc((void**) &srcR_in, sizeof(int) * srcSize));
		checkCudaErrors(hipMalloc((void**) &srcG_in, sizeof(int) * srcSize));
		checkCudaErrors(hipMalloc((void**) &totalR_out, sizeof(double)));
		checkCudaErrors(hipMalloc((void**) &totalG_out, sizeof(double)));

		checkCudaErrors(hipMemcpy(srcR_in, srcR, sizeof(int) * srcSize, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(srcG_in, srcG, sizeof(int) * srcSize, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(totalR_out, totalR, sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(totalG_out, totalG, sizeof(double), hipMemcpyHostToDevice));
	}
	checkCudaErrors(hipMalloc((void**) &srcB_in, sizeof(int) * srcSize));
	checkCudaErrors(hipMalloc((void**) &totalB_out, sizeof(double)));

	checkCudaErrors(hipMemcpy(srcB_in, srcB, sizeof(int) * srcSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(totalB_out, totalB, sizeof(double), hipMemcpyHostToDevice));

	if (isRGB)
		collection <<< dimGrid, dimBlock >>> (
			srcWidth, srcHeight,
			true,
			srcR_in, srcG_in, srcB_in,
			totalR_out, totalG_out, totalB_out
		);
	else
		collection <<< dimGrid, dimBlock >>> (
			srcWidth, srcHeight,
			false,
			NULL, NULL, srcB_in,
			NULL, NULL, totalB_out
		);

	if (isRGB)
	{
		checkCudaErrors(hipMemcpy(totalR, totalR_out, sizeof(double), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(totalG, totalG_out, sizeof(double), hipMemcpyDeviceToHost));
		hipFree(srcR_in);
		hipFree(srcG_in);
		hipFree(totalR_out);
		hipFree(totalG_out);
	}
	checkCudaErrors(hipMemcpy(totalB, totalB_out, sizeof(double), hipMemcpyDeviceToHost));
	hipFree(srcB_in);
	hipFree(totalB_out);
}